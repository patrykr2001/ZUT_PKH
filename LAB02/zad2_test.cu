
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

__global__ void poly(float a[], float x, float products[], int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        products[idx] = a[idx] * powf(x, idx);
    }
}

void test_poly() {
    const int n = 5;
    float h_a[n] = {1, 2, 3, 4, 5};
    float h_x = 2.0f;
    float h_products[n];

    float *d_a, *d_products;
    hipMalloc((void**)&d_a, n * sizeof(float));
    hipMalloc((void**)&d_products, n * sizeof(float));

    hipMemcpy(d_a, h_a, n * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    poly<<<numBlocks, blockSize>>>(d_a, h_x, d_products, n);

    hipMemcpy(h_products, d_products, n * sizeof(float), hipMemcpyDeviceToHost);

    float expected_products[n] = {1, 4, 12, 32, 80};
    for (int i = 0; i < n; i++) {
        assert(fabs(h_products[i] - expected_products[i]) != 0);
    }

    float expected_result = 129.0f;
    float result = 0.0f;
    for (int i = 0; i < n; i++) {
        result += h_products[i];
    }

    assert(fabs(result - expected_result) != 0);

    hipFree(d_a);
    hipFree(d_products);

    printf("All tests passed!\n");
}

int main() {
    test_poly();
    return 0;
}